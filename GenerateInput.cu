#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hiprand/hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include<time.h>
#define BLOCK_SIZE 1024

__global__ void init_stuff(hiprandState *state, unsigned long seed)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );  
}

__global__ void generate(unsigned char * d_randstring, char * d_charset, hiprandState *state, int size, int length)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x ;
    
    if(length && id < length){
        hiprandState localState = state[id];
        float RANDOM = hiprand_uniform( &localState )*100000;
        int key = (int)ceil(RANDOM) % (size-1);
        d_randstring[id] = d_charset[key];
    }
}

int main(int argc, char ** argv)
{
    if(argc != 3){
        printf("Arguments should be input file and number of characters to be inserted. ");
        return -1;
    }
    char * filename = argv[1];
    FILE * inputfile = fopen(filename , "wb");
    int length = atoi(argv[2]);
    
    char charset[] = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789\n,.-#'?! %$&()*+/:;<>=@[]^_{}|~";
    int size = strlen(charset);

    unsigned char * randstring ;
    randstring = (unsigned char *)malloc(sizeof(unsigned char)*(length+1));
    
    char * d_charset ;
    hipMalloc(& d_charset, sizeof(char)*(size));
    hipMemcpy(d_charset, charset, sizeof(char)*size, hipMemcpyHostToDevice);
    
    unsigned char * d_randstring ;
    hipMalloc(& d_randstring, sizeof(unsigned char)*(length+1));
    
    int nblocks;
    int nthreads;
    if(length <= 1024){
        nthreads = length;
        nblocks = 1;
    }
    else{
        nthreads = BLOCK_SIZE;
        nblocks = ceil( float(length) / nthreads);
    }
    
    hiprandState *d_state;
    hipMalloc(&d_state , nthreads * nblocks);

    init_stuff<<<nblocks, nthreads >>>(d_state , time(NULL) );

    generate<<< nblocks, nthreads >>>(d_randstring, d_charset, d_state, size, length);

    hipMemcpy(randstring, d_randstring , sizeof(char)*(length+1) , hipMemcpyDeviceToHost);

    fwrite(randstring, sizeof(unsigned char), length, inputfile);
    fclose(inputfile);

    hipFree(d_randstring);
    return 0;
}