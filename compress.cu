#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <limits.h>
#include <math.h> 
#include <hip/hip_runtime.h>
#include <algorithm>

#define BLOCK_SIZE 1024
__device__ unsigned int counter, counter_2;

__constant__ const unsigned int INTMAX = 2147483647;

__global__ void CalculateFrequency(unsigned char * device_inputFileData , unsigned int * device_frequency, unsigned int inputFileLength)
{
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x ;
    if(id < inputFileLength){
        atomicAdd(& device_frequency[device_inputFileData[id]] , 1);
    }
}

__device__ int findIndex(unsigned int *freq, unsigned int size,unsigned int search){
    for(int i=0;i<size;i++){
        if(freq[i] == search){
            return i;
        }
    }
    return -1;
}
__global__ void findLeastFrequent(unsigned int *freq, unsigned int *min, int size, unsigned int threads, unsigned int* count, unsigned int *index){
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    counter_2 = 0;
    __syncthreads();
    int ind;
    
    if(id<threads){
        
        while(1){
            min[counter_2] = INTMAX;
            
            atomicMin(&min[counter_2], freq[id]);
            // Need global barrier
            __syncthreads();
            
            ind = findIndex(freq, threads, min[counter_2]);
            index[counter_2] = ind;
            // Need global barrier
            __syncthreads();
            freq[ind] = INTMAX;
            
            if(id == 0) atomicInc(&counter_2, size);
            // Need global barrier
            __syncthreads();

            min[counter_2] = INTMAX;
            
            atomicMin(&min[counter_2], freq[id]);
            // Need global barrier
            __syncthreads();
            
            ind = findIndex(freq, threads, min[counter_2]);
            index[counter_2] = ind;
            // Need global barrier
            __syncthreads();
            freq[ind] = min[counter_2] + min[counter_2-1];
            
            if(id == 0) atomicInc(&counter_2, size);
            // Need global barrier
            __syncthreads();
            

            if(min[counter_2] == INTMAX || min[counter_2-1] == INTMAX){
                count[0] = counter_2;
                break;
            }
            
        }
    }
}

int main(int argc, char ** argv){
    unsigned int distinctCharacterCount, inputFileLength;
    unsigned int frequency[256];
    unsigned char * inputFileData, bitSequenceLength = 0, bitSequence[255];
    unsigned int * compressedDataOffset, cpuTimeUsed;
    long unsigned int  memOffset;
    clock_t start, end;

    FILE * inputFile, * compressedFile;

    // check the arguments
    if(argc != 3){
        printf("Arguments should be input file and output file");
        return -1;
    }

    // read input file, get length and data
    inputFile = fopen(argv[1], "rb");
    fseek(inputFile, 0, SEEK_END);
    inputFileLength = ftell(inputFile);
    printf("Input File length : %d\n", inputFileLength);
    fseek(inputFile, 0, SEEK_SET);
    inputFileData = (unsigned char *) malloc(inputFileLength * sizeof(unsigned char));
    fread(inputFileData, sizeof(unsigned char), inputFileLength, inputFile);
    fclose(inputFile);

    // starting the clock, tick tick
    start = clock();

    // find frequency of each symbols
    for(int i = 0; i < 256; i++)
        frequency[i] = 0;

    unsigned int *device_frequency;
    hipMalloc(& device_frequency, 256*sizeof(unsigned int));
    hipMemcpy(device_frequency, frequency, 256*sizeof(unsigned int), hipMemcpyHostToDevice);

    unsigned char * device_inputFileData;
    hipMalloc(& device_inputFileData, inputFileLength*sizeof(unsigned char));
    hipMemcpy(device_inputFileData, inputFileData, inputFileLength*sizeof(unsigned char), hipMemcpyHostToDevice);

    int NumBlocks;
    if( inputFileLength > 1024){
        NumBlocks = ceil( (float)inputFileLength / BLOCK_SIZE );
    }
    else{
        NumBlocks = 1;
    }

    printf("Num of blocks %d\n",NumBlocks);

    CalculateFrequency<<< NumBlocks, BLOCK_SIZE >>>(device_inputFileData, device_frequency, inputFileLength);
    hipMemcpy(frequency, device_frequency, 256*sizeof(unsigned int), hipMemcpyDeviceToHost);

    hipFree(device_inputFileData);
    hipFree(device_frequency);

    // initialize the nodes
    distinctCharacterCount = 0;
    for(int i = 0; i < 256; i++){
        if(frequency[i] > 0){
            distinctCharacterCount ++;
        }
    }

    int unique = 0;
    unsigned char *uniqueChar, *duniqueChar;
    uniqueChar = (unsigned char *)malloc(256*sizeof(unsigned char));
    hipMalloc(&duniqueChar, 256*sizeof(unsigned char));
    for(int i = 0; i<256; i++){
        if(frequency[i] > 0){
            uniqueChar[unique++] = i;
            printf("%d ",frequency[i]);
        }
    }
    printf("\n");
    hipMemcpy(duniqueChar, uniqueChar, 256*sizeof(unsigned char), hipMemcpyHostToDevice);

    // *** FIND MINIMUM 2 FREQUENCY FOR ADDING NEW NODE ***
    unsigned int *tempFreq, *tempDFreq;
    unsigned int *min, *dmin;
    unsigned int *cntMin, *dcntMin;
    unsigned int *indMin, *dindMin;
    int ctr;

    tempFreq = (unsigned int *)malloc(unique*sizeof(unsigned int));
    min = (unsigned int *)malloc(inputFileLength*sizeof(unsigned int));
    cntMin = (unsigned int *)malloc(sizeof(unsigned int));
    indMin = (unsigned int *)malloc(inputFileLength*sizeof(unsigned int));
    ctr = 0;
    for(unsigned int i=0;i<256;i++){
        if(frequency[i]!=0){
            tempFreq[ctr++] = frequency[i];
        }
    }
    // for(unsigned int i=0;i<unique;i++) printf("%d:%c ",tempFreq[i],uniqueChar[i]);
    // printf("\n");
    hipMalloc(&tempDFreq, unique*sizeof(unsigned int));
    hipMalloc(&dmin, inputFileLength*sizeof(unsigned int));
    hipMalloc(&dindMin, inputFileLength*sizeof(unsigned int));
    hipMalloc(&dcntMin, sizeof(unsigned int));
    hipMemcpy(tempDFreq, tempFreq, unique*sizeof(unsigned int), hipMemcpyHostToDevice);

    float num = (float)(unique)/(float)BLOCK_SIZE;
    
    int mod = BLOCK_SIZE;
    if(unique < BLOCK_SIZE) mod = unique%BLOCK_SIZE;
    
    int n = ceil(num);
    printf("%d %d\n",n,mod);
    findLeastFrequent<<<n, mod>>>(tempDFreq, dmin, inputFileLength, unique, dcntMin, dindMin);
    
    hipDeviceSynchronize();

    hipMemcpy(min, dmin, inputFileLength*sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(indMin, dindMin, inputFileLength*sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(cntMin, dcntMin, sizeof(unsigned int), hipMemcpyDeviceToHost);
    // printf("count : %d\n",cntMin[0]);
    // for(unsigned int i=0;i<cntMin[0];i++){
    //     printf("%d:%d:%d ",i,indMin[i],min[i]);
    // } 
    // printf("\n");
    // printf("Min:\n");
    // for(unsigned int i=0;i<cntMin[0];i++) printf("%d ",min[i]);
    // printf("\nIndMin:\n");
    // for(unsigned int i=0;i<cntMin[0];i++) printf("%d ",indMin[i]);

    // end the clock, tick tick
    end = clock();

    cpuTimeUsed = ((end - start)) * 1000 / CLOCKS_PER_SEC;
    printf("\n\nTime taken :: %d:%d s\n", cpuTimeUsed / 1000, cpuTimeUsed % 1000);

    free(inputFileData);

    return 0;
}
